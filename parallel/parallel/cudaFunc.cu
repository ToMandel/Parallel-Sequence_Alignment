#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
// #include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <iostream>
#include <string.h>
#include <limits.h>
#include "cudaHeader.h"

enum constants
{
    RES_INFO = 3,
    MAT_SIZE = 26,
    MAX_SEQ2 = 2000,
};

__device__ int score_mat[MAT_SIZE][MAT_SIZE];
__device__ __constant__ int res_components = 3;

__device__ int cuda_strlen(char* seq)
{
    int counter = 0;
    while(*seq != '\0')
    {
        counter++;
        seq++;
    }

    return counter;
}
char *gen_mutant(char *seq2, int mutant)
{
    char *new_seq = (char *)malloc(sizeof(char) * MAX_SEQ2);
    strcpy(new_seq, seq2);
    for (int i = mutant; i < strlen(seq2); i++)
    {
        if (new_seq[i] >= 'Z')
        {
            new_seq[i] = 'A';
        }
        else
        {
            new_seq[i] = new_seq[i] + 1;
        }
    }
    return new_seq;
}

__global__ void score_offset_mutant(int score_mat[MAT_SIZE][MAT_SIZE],char* seq1, char* seq2 , int* lenght_seq2 , int* start, int* finish, int* res)
{
    int possible_mutants = *lenght_seq2;
    int curr_score = 0;
    int index = 0;

    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    int offset =  thread_id + *start;
    int thread_index_for_res_array = thread_id * res_components; 

    if(offset <= *finish)
    {
        for (int mutant = 0; mutant < possible_mutants; mutant++) //all mutants
        {
            index = offset;
            char *new_seq = gen_mutant(seq2, mutant);
            for (int index_seq2 = 0; index_seq2 < *lenght_seq2; index_seq2++, index++)
            {
                curr_score += score_mat[(seq1[index]) - 'A'][(new_seq[index_seq2]) - 'A'];
            }
            
            if(curr_score > res[thread_index_for_res_array])
            {
                res[thread_index_for_res_array] = curr_score;
                res[thread_index_for_res_array + 1] = offset;
                res[thread_index_for_res_array + 2] = mutant;
            }
            curr_score = 0;
        }
    } 
}

int* cuda_score_offset_mutant(int score_mat[MAT_SIZE][MAT_SIZE],char* seq1, char* seq2 ,int offset_start, int offset_finish)
{
    //CREATE THE ARRAY THAT WILL RETURN THE BEST RESULT
    int* result = (int*)calloc(sizeof(int), RES_INFO);

    //DATA NEEDED
    int offset_size = offset_finish - offset_start + 1; //--> num of cuda threads
    int num_of_blocks = (offset_size / NUM_THREADS_PER_BLOCK);
    if (offset_size % NUM_THREADS_PER_BLOCK != 0)
        num_of_blocks ++;
    
    int lenght_of_res = (RES_INFO) * offset_size; //for each offset the gpu finds the best mutant

    //ALLOCATE DATA TO CUDA MEMORY
    char* cuda_seq1, *cuda_seq2;
    int cuda_matrix[MAT_SIZE][MAT_SIZE];
    int* cuda_offset_start, *cuda_offset_finish ,*cuda_seq2_lenght;
    int* cuda_res, *res = (int*)calloc(sizeof(int) , lenght_of_res);
    int seq2_lenght = (strlen(seq2));

    for (int i = 0; i < lenght_of_res; i++) // get the array ready
    {
        res[i] = INT_MIN;
    }

        //sizes to allocate
    int size_for_cuda_seq1 = sizeof(char) * (strlen(seq1));
    int size_for_cuda_seq2 = sizeof(char) * seq2_lenght;
    int size_for_cuda_matrix = sizeof(int) * (MAT_SIZE*MAT_SIZE);
    int size_for_cuda_int = sizeof(int);
    int size_for_cuda_res = sizeof(int) * lenght_of_res;

        //allocate
    hipMalloc((void**)&cuda_seq1, size_for_cuda_seq1);
    hipMalloc((void**)&cuda_seq2, size_for_cuda_seq2);
    hipMalloc((void**)&cuda_offset_start, size_for_cuda_int);
    hipMalloc((void**)&cuda_offset_finish, size_for_cuda_int);
    hipMalloc((void**)&cuda_res, size_for_cuda_res);
    hipMalloc((void**)&cuda_seq2_lenght, size_for_cuda_int);



    //COPY INPUT INTO DEVICE
    hipMemcpy(cuda_seq1, seq1, size_for_cuda_seq1, hipMemcpyHostToDevice);
    hipMemcpy(cuda_seq2, seq2, size_for_cuda_seq2, hipMemcpyHostToDevice);
    hipMemcpy(cuda_matrix, score_mat, size_for_cuda_matrix, hipMemcpyHostToDevice);
    hipMemcpy(cuda_offset_start, &offset_start, size_for_cuda_int, hipMemcpyHostToDevice);
    hipMemcpy(cuda_offset_finish, &offset_finish, size_for_cuda_int, hipMemcpyHostToDevice);
    hipMemcpy(cuda_seq2_lenght, &seq2_lenght, size_for_cuda_int, hipMemcpyHostToDevice);
    hipMemcpy(cuda_res, res, size_for_cuda_res, hipMemcpyHostToDevice);
    
    
    //LUNCH KERNEL
    score_offset_mutant<<<num_of_blocks, NUM_THREADS_PER_BLOCK>>>(cuda_seq1, cuda_seq2, cuda_seq2_lenght, **cuda_matrix, cuda_offset_start, cuda_offset_finish ,cuda_res);

    //COPY RESULT BACK TO HOST
    hipMemcpy(res, cuda_res, size_for_cuda_res, hipMemcpyDeviceToHost);
    
    //GET THE BIGGEST SCORE
    result[0] = INT_MIN; 

    for (int i = 0; i < lenght_of_res; i += 3)
    {
        if(result[0] < res[i])
        {
            result[0] = res[i];
            result[1] = res[i+1];
            result[2] = res[i+2];
        }
    }
    
    //FREE
    hipFree(cuda_seq1);
    hipFree(cuda_seq2);
    hipFree(cuda_matrix);
    hipFree(cuda_offset_start);
    hipFree(cuda_offset_finish);
    hipFree(cuda_res);
    hipFree(cuda_seq2_lenght);

    return result;
}
